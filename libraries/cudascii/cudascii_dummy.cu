#include <hip/hip_runtime.h>

#include <chrono>
#include <cmath>
#include <iostream>
#include <string>
#include <thread>

namespace {
    // Algorithm Parameterization
    // const std::string gray_levels_fine =
    // "$@B%8&WM#*oahkbdpqwmZO0QLCJUYXzcvunxrjft/\\|()1{}[]?-_+~i!lI;:,\"^`. ";
    const constexpr char* gray_level_lookup{"@%#*+=-:. "};
    const constexpr int gray_levels = 10;
    const constexpr float RED_WEIGHT = 0.2126;
    const constexpr float GREEN_WEIGHT = 0.7152;
    const constexpr float BLUE_WEIGHT = 0.0722;
    const constexpr float CONVERSION_THRESHOLD = 0.0031308;
    const constexpr float BELOW_THRESHOLD_SCALAR = 12.92;
    const constexpr float ABOVE_THRESHOLD_SCALAR = 1.055;
    const constexpr float ABOVE_THRESHOLD_EXPONENT = 1 / 2.4;
    const constexpr float ABOVE_THRESHOLD_OFFSET = -0.055;
}

namespace cudascii {

    __global__ void pixel_to_ascii(unsigned char *out, unsigned char r, unsigned char g, unsigned char b) {

        // Thread index
        int i = threadIdx.x + blockIdx.x * blockDim.x;

        float c_linear, c_srgb;
        int gray_index;
        
        // Standard linear combination
        c_linear = RED_WEIGHT*(r/255.) + GREEN_WEIGHT*(g/255.) + BLUE_WEIGHT*(b/255.);
        
        // If gray level is very dark, use linear scaling
        if (c_linear <= CONVERSION_THRESHOLD)
            c_srgb = BELOW_THRESHOLD_SCALAR * c_linear;

        // Non linear scaling to adjust for gamma exposure
        else
            c_srgb = ABOVE_THRESHOLD_SCALAR * powf(c_linear,ABOVE_THRESHOLD_EXPONENT) + ABOVE_THRESHOLD_OFFSET;
        
        // Scale c_srgb to the gray levels while handling an edge case of c_srgb = 1
        gray_index = (int) fmin(c_srgb * gray_levels, gray_levels - 1.);

        // Final character representing the gray level of the RGB pixel
        out[i] = gray_level_lookup[gray_index];
    }


    __global__ void set_high(unsigned char *out) {

        // Thread index
        int i = threadIdx.x + blockIdx.x * blockDim.x;
        out[i] = 255;

    }

    bool test_cuda() {

        // Assess how much memory is needed for image
        const unsigned int N = 1'000'000'000;
        const unsigned int bytes = N * sizeof(unsigned char);

        // Allocate GPU memory
        unsigned char *d_a;
        if(hipMalloc((unsigned char**)&d_a, bytes) != hipSuccess)
        {
            std::cout << "failed!" << std::endl;
            return false;
        }
        else
        {
            std::cout << "passed!" << std::endl;
        }

        // Copy the image from host (CPU) to device (GPU)
        // cudaMemcpy(d_a, src, bytes, cudaMemcpyHostToDevice);

        // Call the kernel code here
        int threadsPerBlock = 256;
        int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
        set_high<<<threadsPerBlock,blocksPerGrid>>>(d_a);

        // Copy the ascii array from device (GPU) to host (CPU)
        // cudaMemcpy(h_a, d_a, bytes, cudaMemcpyDeviceToHost);

        using namespace std::chrono_literals;
        std::this_thread::sleep_for(3s);

        // Don't forget to free memory!!!!
        hipFree(d_a);
        
        return true;

    }
}